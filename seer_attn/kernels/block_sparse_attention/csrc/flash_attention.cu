#include "hip/hip_runtime.h"
#include "attention_api.cuh"
#include <cassert>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <cutlass/numeric_conversion.h>
#include <cutlass/numeric_types.h>
#include <torch/extension.h>
#include <torch/python.h>
#include <vector>

#include "static_switch.h"
#include "kernel_traits.h"
#include "flash.h"
#include "utils.h"

namespace flash {

using namespace cute;

template <int kBlockM, int kBlockN, int kNWarps,typename Engine, typename Layout>
inline __device__ void mask_within_nblock(Tensor<Engine, Layout> &tensor, const int m_block, const int nbi) {
    // tensor has shape (nrow=(2, MMA_M), ncol=(2, MMA_N))
    static_assert(Layout::rank == 2, "Only support 2D Tensor");
    // NOTE:
    // Determining the index within an MMA is also a challenge
    // (nrow=(2, MMA_M), ncol=(2, MMA_N)) looks like:
    //    T1.V0 T1.V1
    //    T1.V0 T1.V1
    // Determine col and row values based on the mma_tile diagram

    // NOTE:
    // Calculate the processing range of the thread, mask out the parts beyond the range
    //
    // NOTE:
    // % 32 means grouping by 32, because the maximum thread id in SM80_16x8x16_F32F16F16F32_TN _1_2_1 is 32
    // (lane_id % 4) * 2 indicates which "color" col(thread) it is in, *2 is to align to the right (i.e., which value2 is being processed)
    // Therefore, col_idx_offset represents which column in the 4 columns of the single Atom the current thread is processing

    // lane_id represents a "thread group" in an MMA tile
    const int lane_id = threadIdx.x % 32;
    const int col_idx_offset = kBlockN * nbi + (lane_id % 4) * 2;

    const int nrow_group = threadIdx.x / 32;
    const int row_idx_offset = kBlockM * m_block + lane_id / 4 + nrow_group * 16 /* 2*8 */;
    // (2, nrow), 2*8 for each
    const int group_stride = kNWarps * 16;

    #pragma unroll
    for (int nj = 0; nj < size<1, 1>(tensor); ++nj) {
        // In SM80_16x8x16_F32F16F16F32_TN, a group of 4 threads processes 8 values in a row
        const int col_idx_base = col_idx_offset + nj * 8;
        #pragma unroll
        for (int j = 0; j < size<1, 0>(tensor); ++j) {
            // j is used to calculate the col for value 1 and value 2
            // col_idx ultimately represents the column number of the value being processed by the current thread
            const int col_idx = col_idx_base + j;

            // Mask out the parts of the scores (result after QK) that are beyond the range
            // Compare column and row numbers

            // Without the "make_coord" we get wrong results
            // for nrow(2, MMA_M)
            #pragma unroll
            for (int mi = 0; mi < size<0, 0>(tensor); ++mi) {

              #pragma unroll
              for (int mj = 0; mj < size<0, 1>(tensor); ++mj) {
                const int row_idx = row_idx_offset + mi * 8 + mj * group_stride;
                if (col_idx > row_idx) {
                  tensor(make_coord(mi, mj), make_coord(j, nj)) = -INFINITY;
                }
              }

            }

        }
    }
}

// NOTE: GEMM encapsulation with matrix A already in registers
template<typename Tensor0, typename Tensor1, typename Tensor2, typename Tensor3,
         typename TiledMma, typename TiledCopy, typename ThrCopy>
inline __device__ void gemm_A_in_regs(Tensor0 &acc, Tensor1 &tCrA, Tensor2 &tCrB, Tensor3 const& tCsB,
                                      TiledMma tiled_mma, TiledCopy smem_tiled_copy_B,
                                      ThrCopy smem_thr_copy_B) {
    // NOTE: Conforms to M N K description: A[M, K] @ B[N, K] = C[M, N]
    CUTE_STATIC_ASSERT_V(size<1>(tCrA) == size<1>(acc));                     // MMA_M
    CUTE_STATIC_ASSERT_V(size<1>(tCrB) == size<2>(acc));                     // MMA_N
    CUTE_STATIC_ASSERT_V(size<2>(tCrA) == size<2>(tCrB));                     // MMA_K
    // NOTE: Retile to the size needed for copying
    Tensor tCrB_copy_view = smem_thr_copy_B.retile_D(tCrB);
    CUTE_STATIC_ASSERT_V(size<1>(tCsB) == size<1>(tCrB_copy_view));            // N
    cute::copy(smem_tiled_copy_B, tCsB(_, _, _0{}), tCrB_copy_view(_, _, _0{}));
    #pragma unroll
    for (int i = 0; i < size<2>(tCrA); ++i) {
        if (i < size<2>(tCrA) - 1) {
            cute::copy(smem_tiled_copy_B, tCsB(_, _, i + 1), tCrB_copy_view(_, _, i + 1));
        }
        cute::gemm(tiled_mma, tCrA(_, _, i), tCrB(_, _, i), acc);
    }
}

template<typename Tensor0, typename Tensor1,
         typename Tensor2, typename Tensor3, typename Tensor4,
         typename TiledMma, typename TiledCopyA, typename TiledCopyB,
         typename ThrCopyA, typename ThrCopyB>
inline __device__ void gemm_smem(Tensor0 &acc, Tensor1 &tCrA, Tensor2 &tCrB, Tensor3 const& tCsA,
                            Tensor4 const& tCsB, TiledMma tiled_mma,
                            TiledCopyA smem_tiled_copy_A, TiledCopyB smem_tiled_copy_B,
                            ThrCopyA smem_thr_copy_A, ThrCopyB smem_thr_copy_B) {
    CUTE_STATIC_ASSERT_V(size<1>(tCrA) == size<1>(acc));                     // MMA_M
    CUTE_STATIC_ASSERT_V(size<1>(tCrB) == size<2>(acc));                     // MMA_N
    CUTE_STATIC_ASSERT_V(size<2>(tCrA) == size<2>(tCrB));                     // MMA_K
    Tensor tCrA_copy_view = smem_thr_copy_A.retile_D(tCrA);
    CUTE_STATIC_ASSERT_V(size<1>(tCsA) == size<1>(tCrA_copy_view));            // M
    Tensor tCrB_copy_view = smem_thr_copy_B.retile_D(tCrB);
    CUTE_STATIC_ASSERT_V(size<1>(tCsB) == size<1>(tCrB_copy_view));            // N
    // NOTE: s -> reg
    cute::copy(smem_tiled_copy_A, tCsA(_, _, _0{}), tCrA_copy_view(_, _, _0{}));
    cute::copy(smem_tiled_copy_B, tCsB(_, _, _0{}), tCrB_copy_view(_, _, _0{}));
    #pragma unroll
    for (int i = 0; i < size<2>(tCrA); ++i) {
        if (i < size<2>(tCrA) - 1) {
            cute::copy(smem_tiled_copy_A, tCsA(_, _, i + 1), tCrA_copy_view(_, _, i + 1));
            cute::copy(smem_tiled_copy_B, tCsB(_, _, i + 1), tCrB_copy_view(_, _, i + 1));
        }
        cute::gemm(tiled_mma, tCrA(_, _, i), tCrB(_, _, i), acc);
    }
}

// Blocks until all but N previous cp.async.commit_group operations have committed.
// This differs from cute::cp_async_wait in that when N = 0 we don't call cp.async.wait_all
// (which is equivalent to commit_group then wait_group 0).
// Instead we just call cp.async.wait_group 0, which is slightly faster.
// https://github.com/NVIDIA/cutlass/blob/master/include/cute/arch/copy_sm80.hpp#L113
template <int N>
CUTE_HOST_DEVICE
void cp_async_wait() {
#if defined(CUTE_ARCH_CP_ASYNC_SM80_ENABLED)
    asm volatile("cp.async.wait_group %0;\n" :: "n"(N));
#endif
}

// copy from S to D with tiled_copy
// TODO: Need to support skipping copy in causal mode
template <typename TiledCopy, typename Engine0, typename Layout0, typename Engine1, typename Layout1>
inline __device__ void copy(TiledCopy tiled_copy, Tensor<Engine0, Layout0> const &S,
                            Tensor<Engine1, Layout1> &D) {
    CUTE_STATIC_ASSERT_V(rank(S) == Int<3>{});
    CUTE_STATIC_ASSERT_V(rank(D) == Int<3>{});
    CUTE_STATIC_ASSERT_V(size<0>(S) == size<0>(D));                     // MMA
    CUTE_STATIC_ASSERT_V(size<1>(S) == size<1>(D));                     // MMA_M
    CUTE_STATIC_ASSERT_V(size<2>(S) == size<2>(D));                     // MMA_K

    #pragma unroll
    for (int m = 0; m < size<1>(S); ++m) {
        // TODO: In the original version, identity_MN is used to skip large blocks, predicate is used to skip copying within the block
        // TODO: Add predicate logic to skip unnecessary copying
        // if (get<0>(identity_MN(0, m, 0)) < max_MN)
        #pragma unroll
        for (int k = 0; k < size<2>(S); ++k) {
          cute::copy(tiled_copy, S(_, m, k), D(_, m, k));
        }
    }
}


// Convert rowcol_layout from (nrow=(2, MMA_M), ncol=(2, MMA_N)) to ((2, 2, 2), MMA_M, MMA_N / 2)
// if using m16n8k16, or to ((2, 2, 1), MMA_M, MMA_N) if using m16n8k8.
template<typename MMA_traits, typename Layout>
inline __device__ auto convert_layout_rowcol_Aregs(Layout rowcol_layout) {
    using X = Underscore;
    static_assert(decltype(size<0, 0>(rowcol_layout))::value == 2);
    static_assert(decltype(size<1, 0>(rowcol_layout))::value == 2);
    constexpr int mma_shape_K = get<2>(typename MMA_traits::Shape_MNK{});
    static_assert(mma_shape_K == 8 || mma_shape_K == 16);
    constexpr int MMA_N_divisor = mma_shape_K == 8 ? 1 : 2;
    auto l = logical_divide(rowcol_layout, Shape<X, Shape<X, Int<MMA_N_divisor>>>{});  // ((2, MMA_M), (2, (2, MMA_N / 2)))
    // TD [2023-08-13]: Same error as above on Cutlass 3.2
    // return make_layout(make_layout(get<1, 0>(l), get<0, 0>(l), get<1, 1, 0>(l)),
    //                    get<0, 1>(l),
    //                    get<1, 1, 1>(l));
    return make_layout(make_layout(get<0>(get<1>(l)), get<0>(get<0>(l)), get<0>(get<1>(get<1>(l)))),
                       get<1>(get<0>(l)),
                       get<1>(get<1>(get<1>(l))));
};


// TODO: not work
template <typename To_type, typename Engine, typename Layout>
inline __device__ auto convert_type(Tensor<Engine, Layout> const &tensor) {
    using From_type = typename Engine::value_type;
    constexpr int numel = decltype(size(tensor))::value;
    cutlass::NumericArrayConverter<To_type, From_type, numel> convert_op;
    // HACK: this requires tensor to be "contiguous"
    auto frag = convert_op(*reinterpret_cast<const cutlass::Array<From_type, numel> *>(tensor.data()));
    return make_tensor(make_rmem_ptr<To_type>(&frag), tensor.layout());
}


template <typename Fragment>
inline __device__ auto convert_type_f32_to_f16(Fragment const &acc_fp32) {
  Tensor acc_fp16 = make_tensor<cute::half_t>(shape(acc_fp32));
  {
    Tensor acc_fp32x2 = recast< float2>(acc_fp32);
    Tensor acc_fp16x2 = recast<__half2>(acc_fp16);
    for (int i = 0; i < size(acc_fp32x2); ++i) { acc_fp16x2(i) = __float22half2_rn(acc_fp32x2(i)); }
  }
  return acc_fp16;
}

// Apply the exp to all the elements.
template <bool Scale_max=true, typename Engine0, typename Layout0, typename Engine1, typename Layout1>
inline __device__ void scale_apply_exp2(Tensor<Engine0, Layout0> &tensor, Tensor<Engine1, Layout1> const &max, const float scale) {
    static_assert(Layout0::rank == 2, "Only support 2D Tensor");
    static_assert(Layout1::rank == 1, "Only support 1D Tensor");
    CUTE_STATIC_ASSERT_V(size<0>(max) == size<0>(tensor));
    #pragma unroll
    for (int mi = 0; mi < size<0>(tensor); ++mi) {
        // If max is -inf, then all elements must have been -inf (possibly due to masking).
        // We don't want (-inf - (-inf)) since that would give NaN.
        // If we don't have float around M_LOG2E the multiplication is done in fp64.
        const float max_scaled = max(mi) == -INFINITY ? 0.f : max(mi) * (Scale_max ? scale : float(M_LOG2E));
        #pragma unroll
        for (int ni = 0; ni < size<1>(tensor); ++ni)  {
            // Instead of computing exp(x - max), we compute exp2(x * log_2(e) -
            // max * log_2(e)) This allows the compiler to use the ffma
            // instruction instead of fadd and fmul separately.
            tensor(mi, ni) = expf(tensor(mi, ni) * scale - max_scaled);
        }
    }
}

// Convert acc_layout from (MMA=4, MMA_M, MMA_N) to (nrow=(2, MMA_M), ncol=(2, MMA_N))
// TODO: Understand the mathematical meaning after convert_layout_acc_rowcol
// A visual explanation is that it transforms:
//    T1.V0
//    T1.V1
//    T1.V0
//    T1.V1
// into:
//    T1.V0 T1.V1
//    T1.V0 T1.V1
// This aligns with the row-column intuition of the MMA tile.
template<typename Layout>
inline __device__ auto convert_layout_acc_rowcol(Layout acc_layout) {
    static_assert(decltype(size<0>(acc_layout))::value == 4);
    static_assert(decltype(rank(acc_layout))::value == 3);
    auto l = logical_divide(acc_layout, Shape<_2>{});  // ((2, 2), MMA_M, MMA_N)
    // TD [2023-08-13]: Idk why but get<0, 1>(l) doesn't work for Cutlass 3.2, I'm getting
    // "int_tuple.hpp(74): error: conversion to inaccessible base class"
    // return make_layout(make_layout(get<0, 1>(l), get<1>(l)), make_layout(get<0, 0>(l), get<2>(l)));
    return make_layout(make_layout(get<1>(get<0>(l)), get<1>(l)), make_layout(get<0>(get<0>(l)), get<2>(l)));
};

template<bool Is_first, typename Tensor0, typename Tensor1, typename Tensor2>
inline __device__ void softmax_rescale_o(Tensor0 &scores, Tensor1 &scores_max, Tensor1 &scores_sum,
                                         Tensor2 &acc_o, float softmax_scale_log2) {
    // NOTE: scores come from acc_s: Q@K.T
    // acc_s is used to store the result of QK and softmax [seqlen, seqlen]
    // acc_o is used to store the numerator part of the softmax(QK) result, for rescaling
    // Streaming computation continuously rescales with the current block computation result scores

    if (Is_first) {
        // NOTE: Optimization, the first softmax does not need rescaling, only needs to record the numerator, max, sum
        reduce_max</*zero_init=*/true>(scores, scores_max);
        flash::scale_apply_exp2(scores, scores_max, softmax_scale_log2);
        reduce_sum(scores, scores_sum);
    } else {
        // Record the previous max
        Tensor scores_max_prev = make_fragment_like(scores_max);
        cute::copy(scores_max, scores_max_prev);
        // TODO: Learn the implementation of reduce
        // NOTE: Calculate the new max into scores_max
        // reduce_max includes steps:
        //  1. Calculate the max within the current thread: iterate
        //  2. Reduce the max across threads: use shift trick to reduce
        reduce_max</*zero_init=*/false>(scores, scores_max);
        // Reshape acc_o from (MMA=4, MMA_M, MMA_K) to (nrow=(2, MMA_M), ncol=(2, MMA_K))
        // Convert acc_o into a shape that aligns with 2D intuition (nrow, ncol)
        Tensor acc_o_rowcol = make_tensor(acc_o.data(), flash::convert_layout_acc_rowcol(acc_o.layout()));
        #pragma unroll
        for (int mi = 0; mi < size(scores_max); ++mi) {
            // NOTE: Auxiliary variable: current max
            float scores_max_cur = scores_max(mi);
            // NOTE: Calculate the rescale value for the old score
            // NOTE: Since QK (affecting max) was calculated without considering softmax_scale, we need to compensate here
            float scores_scale = expf((scores_max_prev(mi) - scores_max_cur) * softmax_scale_log2);
            // NOTE: Rescale the old denominator part
            scores_sum(mi) *= scores_scale;
            // NOTE: Rescale the old numerator part
            // acc_o_rowcol.shape = (nrow, ncol)
            #pragma unroll
            for (int ni = 0; ni < size<1>(acc_o_rowcol); ++ni) { acc_o_rowcol(mi, ni) *= scores_scale; }
        }
        // NOTE: Calculate the new numerator part: rescale all scores
        flash::scale_apply_exp2(scores, scores_max, softmax_scale_log2);

        // NOTE: Accumulate the new denominator
        Tensor scores_sum_cur = make_fragment_like(scores_sum);
        // NOTE: Use the new numerator to accumulate the new denominator
        //  1. Accumulate within the thread: iterate
        //  2. Accumulate across threads: use shift trick to reduce
        reduce_sum(scores, scores_sum_cur);
        // NOTE: Add the new denominator to the old denominator
        #pragma unroll
        for (int mi = 0; mi < size(scores_sum); ++mi) { scores_sum(mi) += scores_sum_cur(mi); }
    }
};

} // namespace flash

void set_params_fprop(Flash_fwd_params &params,

                      // device pointers
                      const torch::Tensor q,
                      const torch::Tensor k,
                      const torch::Tensor v,
                      torch::Tensor out,

                      void *softmax_lse_d,
                      float softmax_scale,
                      bool is_causal) {

  memset(&params, 0, sizeof(params));

  params.bs = q.size(0);
  params.head = q.size(1);
  params.q_seqlen = q.size(2);
  params.dim = q.size(3);

  params.k_head = k.size(1);
  params.k_seqlen = k.size(2);

  params.bs_stride = q.stride(0);
  params.head_stride = q.stride(1);
  params.seqlen_stride = q.stride(2);
  params.dim_stride = q.stride(3);

  params.softmax_scale = softmax_scale;
  // TODO: Use log2 for scaling
  params.softmax_scale_log2 = softmax_scale * M_LOG2E;
  params.is_causal = is_causal;
  params.is_bf16 = q.dtype() == torch::kBFloat16;

  // LogSumExp save for backward
  params.softmax_lse_ptr = softmax_lse_d;

  // TODO: get ptr
  params.q_ptr = q.data_ptr();
  params.k_ptr = k.data_ptr();
  params.v_ptr = v.data_ptr();
  params.out_ptr = out.data_ptr();
}


// Shared Storage with Aligned addresses.
template <class ElementType, class SmemLayoutQ, class SmemLayoutK, class SmemLayoutV>
struct SharedStorage {
  // TODO: If aligned, does smem calculation have issues?
  cute::array_aligned<ElementType, cute::cosize_v<SmemLayoutQ>> smem_q;
  cute::array_aligned<ElementType, cute::cosize_v<SmemLayoutK>> smem_k;
  cute::array_aligned<ElementType, cute::cosize_v<SmemLayoutV>> smem_v;
};

template <typename Kernel_traits, bool Is_causal=false, typename Params>
__global__ void flash_attention_v2_cutlass_kernel(const Params params) {

  using namespace cute;

  // m block index
  const int m_block = blockIdx.x;

  // bs * head
  const int base_id = blockIdx.y;
  // The thread index.
  const int tidx = threadIdx.x;

  // TODO: Pass in generics
  // NOTE: Small trick
  using Element = typename Kernel_traits::Element;
  using ElementAccum = typename Kernel_traits::ElementAccum;
  // using TiledMMA = typename Kernel_traits::MMA;
  using TiledMMA = typename Kernel_traits::TiledMma;
  using index_t = typename Kernel_traits::index_t;
  using SmemLayoutQ = typename Kernel_traits::SmemLayoutQ;
  using SmemLayoutK = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutV = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutVt = typename Kernel_traits::SmemLayoutVtransposed;
  using SmemLayoutVtNoSwizzle = typename Kernel_traits::SmemLayoutVtransposedNoSwizzle;

  constexpr int kNWarps = Kernel_traits::kNWarps;
  constexpr int kBlockM = Kernel_traits::kBlockM;
  constexpr int kBlockN = Kernel_traits::kBlockN;
  constexpr int kHeadDim = Kernel_traits::kHeadDim;

  // Shared memory.
  extern __shared__ char smem_[];
  using SharedStorage = SharedStorage<Element, SmemLayoutQ, SmemLayoutK, SmemLayoutV>;
  SharedStorage &shared_storage = *reinterpret_cast<SharedStorage *>(smem_);

  const int bs_head_offset = base_id * params.head_stride;

  // TODO: base offset for MHA
  // NOTE: convert C pointer to Tensor for convenience
  Tensor Q = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.q_ptr) + bs_head_offset),
      make_shape(params.q_seqlen, Int<kHeadDim>{}),
      make_stride(Int<kHeadDim>{}, Int<1>{}));
  Tensor K = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.k_ptr) + bs_head_offset),
      make_shape(params.k_seqlen, Int<kHeadDim>{}),
      make_stride(Int<kHeadDim>{}, Int<1>{}));
  Tensor V = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.v_ptr) + bs_head_offset),
      make_shape(params.k_seqlen, Int<kHeadDim>{}),
      make_stride(Int<kHeadDim>{}, Int<1>{}));
  Tensor O = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.out_ptr) + bs_head_offset),
      make_shape(params.q_seqlen, Int<kHeadDim>{}),
      make_stride(Int<kHeadDim>{}, Int<1>{}));
  // TODO:
  Tensor LSE = make_tensor(
      make_gmem_ptr(reinterpret_cast<ElementAccum *>(params.softmax_lse_ptr) + base_id * params.q_seqlen),
      // Shape<Int<kBlockM>, Stride<_1>{}>{}, 
      make_shape(params.q_seqlen),
      make_stride(Int<1>{}));


  // Load Q, K, V blocks
  // (kBlockM, kHeadDim, num_tile_n)
  Tensor gQ = local_tile(Q, make_tile(Int<kBlockM>{}, Int<kHeadDim>{}), make_coord(m_block, _));

  // (kBlockN, kHeadDim, num_tile_n)
  // NOTE: Loading pipeline, initial load of required K, V
  Tensor gK = local_tile(K, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(0, _));
  Tensor gV = local_tile(V, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(0, _));

  // Get MMA abstraction
  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(tidx);

  // Construct SMEM tensors.
  Tensor sQ = make_tensor(make_smem_ptr(shared_storage.smem_q.data()), SmemLayoutQ{});
  Tensor sK = make_tensor(make_smem_ptr(shared_storage.smem_k.data()), SmemLayoutK{});
  Tensor sV = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutV{});

  // Tensor for V Transpose; used in GEMM-II.
  Tensor sVt = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutVt{});
  Tensor sVtNoSwizzle = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutVtNoSwizzle{});

  // NOTE: Copy abstraction
  // NOTE: QKV gmem -> smem copy abstraction
  typename Kernel_traits::GmemTiledCopyQKV gmem_tiled_copy_QKV;
  auto gmem_thr_copy_QKV = gmem_tiled_copy_QKV.get_thread_slice(tidx);

  // NOTE: Define src, dst for gmem -> smem copy
  Tensor tQgQ = gmem_thr_copy_QKV.partition_S(gQ(_, _, 0));
  Tensor tQsQ = gmem_thr_copy_QKV.partition_D(sQ);
  Tensor tKgK = gmem_thr_copy_QKV.partition_S(gK(_, _, 0));
  Tensor tKsK = gmem_thr_copy_QKV.partition_D(sK);
  Tensor tVgV = gmem_thr_copy_QKV.partition_S(gV(_, _, 0));
  Tensor tVsV = gmem_thr_copy_QKV.partition_D(sV);


  // NOTE: Define dst for smem -> reg copy
  // partition_fragment is similar to partition, but returns a register representation
  Tensor tSrQ  = thr_mma.partition_fragment_A(sQ);                           // (MMA,MMA_M,MMA_K)
  Tensor tSrK  = thr_mma.partition_fragment_B(sK);                           // (MMA,MMA_N,MMA_K)
  Tensor tOrVt  = thr_mma.partition_fragment_B(sVtNoSwizzle);                // (MMA, MMA_K,MMA_N)

  //
  // Copy Atom retiling
  //

  // TODO: Understand the atom retiling here

  // NOTE: Prepare copy objects for Q, K, V to smem
  auto smem_tiled_copy_Q = make_tiled_copy_A(typename Kernel_traits::SmemCopyAtom{}, tiled_mma);
  auto smem_thr_copy_Q = smem_tiled_copy_Q.get_thread_slice(tidx);
  Tensor tSsQ = smem_thr_copy_Q.partition_S(sQ);

  auto smem_tiled_copy_K = make_tiled_copy_B(typename Kernel_traits::SmemCopyAtom{}, tiled_mma);
  auto smem_thr_copy_K = smem_tiled_copy_K.get_thread_slice(tidx);
  Tensor tSsK = smem_thr_copy_K.partition_S(sK);

  // TODO: Transpose during copy
  // NOTE: smem->reg copy Vt
  auto smem_tiled_copy_V = make_tiled_copy_B(typename Kernel_traits::SmemCopyAtomTransposed{}, tiled_mma);
  auto smem_thr_copy_V = smem_tiled_copy_V.get_thread_slice(tidx);
  Tensor tOsVt = smem_thr_copy_V.partition_S(sVt);

  // Pipeline loading initial Q, K
  // Load Q to smem
  flash::copy(gmem_tiled_copy_QKV, tQgQ, tQsQ);
  // Load K to smem
  flash::copy(gmem_tiled_copy_QKV, tKgK, tKsK);
  // Start async copy
  cute::cp_async_fence();

  Tensor rAccOut = partition_fragment_C(tiled_mma, Shape<Int<kBlockM>, Int<kHeadDim>>{});

  // step1: slice-k compute QK block
  // Q[BLOCK_M, BLOCK_N] @ K[BLOCK_M, BLOCK_N].T = O[BLOCK_M, BLOCK_M]
  //
  // step2:
  // advance K, V

  // NOTE: Number of K, V blocks: processing range
  const int n_block_min = 0;
  // NOTE: 1. mask between N BLOCKs if is causal mode
  int seqlen_start = m_block * kBlockM;
  int seqlen_end = (m_block + 1) * kBlockM;
  int n_block_max = Is_causal ? cute::ceil_div(seqlen_end, kBlockN) : cute::ceil_div(params.k_seqlen, kBlockN);

  // NOTE: Max to be recorded
  Tensor scores_max = make_tensor<ElementAccum>(Shape<Int<2 * size<1>(rAccOut)>>{});
  // NOTE: Denominator to be recorded
  Tensor scores_sum = make_fragment_like(scores_max);

  clear(rAccOut);

  for (int nbi = n_block_min; nbi < n_block_max; nbi++) {
    auto rAccScore = partition_fragment_C(tiled_mma, make_shape(Int<kBlockM>{}, Int<kBlockN>{}));

    clear(rAccScore);

    // Wait for Q, K gmem -> smem copy to complete, i.e., Q, K ready
    // wait<0> means wait for 0 remaining
    flash::cp_async_wait<0>();
    __syncthreads();

    // Asynchronously load V while doing gemm
    gV = local_tile(V, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(nbi, _));
    tVgV = gmem_thr_copy_QKV.partition_S(gV(_, _, 0));
    // Asynchronously load V to smem
    flash::copy(gmem_tiled_copy_QKV, tVgV, tVsV);
    // Initiate async copy
    cute::cp_async_fence();


    // O = Q@K.T
    // NOTE: Load data from smem into registers before performing GEMM, **retile during loading**
    flash::gemm_smem(rAccScore, tSrQ, tSrK, tSsQ, tSsK, tiled_mma, smem_tiled_copy_Q, smem_tiled_copy_K,
        smem_thr_copy_Q, smem_thr_copy_K
    );

    Tensor scores = make_tensor(rAccScore.data(), flash::convert_layout_acc_rowcol(rAccScore.layout()));

    // NOTE: 2. Mask within N BLOCKs
    if (Is_causal ==  true && nbi * kBlockN >= seqlen_start) {
      flash::mask_within_nblock<kBlockM, kBlockN, kNWarps>(scores, m_block, nbi);
    }

    // NOTE: Wait for V to finish loading, prepare the initial state for the next K load
    flash::cp_async_wait<0>();
    __syncthreads();

    // Advance K
    if (nbi != n_block_max - 1) {
      gK = local_tile(K, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(nbi + 1, _));
      tKgK = gmem_thr_copy_QKV.partition_S(gK(_, _, 0));
      flash::copy(gmem_tiled_copy_QKV, tKgK, tKsK);
      cute::cp_async_fence();
    }

    // Compute softmax
    // NOTE: rAccOut records all numerators after softmax
    nbi == 0 ? flash::softmax_rescale_o</*Is_first=*/true>(scores, scores_max, scores_sum, rAccOut, params.softmax_scale) :
      flash::softmax_rescale_o</*Is_first=*/false>(scores, scores_max, scores_sum, rAccOut, params.softmax_scale);

    // Perform QK @ V computation
    // (score AKA rAccScore): QK[M, N] @ V[N, dim]
    // NOTE: DABC: F32F16F16F32, convert D type (F32) to A type (F16)
    // TODO: convert_type is currently hardcoded
    Tensor rP = flash::convert_type_f32_to_f16(rAccScore);
    // NOTE: Convert from layout C to layout A
    Tensor tOrP = make_tensor(rP.data(), flash::convert_layout_rowcol_Aregs<TiledMMA>(scores.layout()));

    flash::gemm_A_in_regs(rAccOut, tOrP, tOrVt, tOsVt, tiled_mma, smem_tiled_copy_V, smem_thr_copy_V);
  }

  // Epilogue

  // NOTE: Finally, divide by the denominator
  // Reshape acc_o from (MMA=4, MMA_M, MMA_K) to (nrow=(2, MMA_M), ncol=(2, MMA_K))
  // AKA reshape to (nrow, ncol) but with specific MMA layout
  Tensor acc_o_rowcol = make_tensor(rAccOut.data(), flash::convert_layout_acc_rowcol(rAccOut.layout()));
  // NOTE: Save lse for backward pass
  Tensor lse = make_fragment_like(scores_sum);
  // For row
  #pragma unroll
  for (int mi = 0; mi < size<0>(acc_o_rowcol); ++mi) {
    float sum = scores_sum(mi);
    float inv_sum = (sum == 0.f || sum != sum) ? 1.f : 1.f / sum;
    // Compute lse
    // NOTE: Here we use max * scale
    lse(mi) = (sum == 0.f || sum != sum) ? INFINITY : scores_max(mi) * params.softmax_scale + __logf(sum);
    float scale = inv_sum;
    // For col
    #pragma unroll
    for (int ni = 0; ni < size<1>(acc_o_rowcol); ++ni) {
      acc_o_rowcol(mi, ni) *= scale;
    }
  }

  // Convert acc_o from fp32 to fp16/bf16
  Tensor rO = flash::convert_type_f32_to_f16(rAccOut);
  // Reuse sQ's smem for copying out sO
  Tensor sO = make_tensor(sQ.data(), typename Kernel_traits::SmemLayoutO{});    // (SMEM_M,SMEM_N)

  // Partition sO to match the accumulator partitioning
  // TODO: Review
  auto smem_tiled_copy_O = make_tiled_copy_C(typename Kernel_traits::SmemCopyAtomO{}, tiled_mma);
  auto smem_thr_copy_O = smem_tiled_copy_O.get_thread_slice(tidx);
  Tensor taccOrO = smem_thr_copy_O.retile_S(rO);        // ((Atom,AtomNum), MMA_M, MMA_N)
  Tensor taccOsO = smem_thr_copy_O.partition_D(sO);     // ((Atom,AtomNum),PIPE_M,PIPE_N)

  // NOTE: Copy to smem first
  cute::copy(smem_tiled_copy_O, taccOrO, taccOsO);

  Tensor gO = local_tile(O, make_tile(Int<kBlockM>{}, Int<kHeadDim>{}), make_coord(m_block, _));

  // Create copy from smem -> gmem
  typename Kernel_traits::GmemTiledCopyO gmem_tiled_copy_O;
  auto gmem_thr_copy_O = gmem_tiled_copy_O.get_thread_slice(tidx);
  Tensor tOsO = gmem_thr_copy_O.partition_S(sO);        // ((Atom,AtomNum),ATOM_M,ATOM_N)
  Tensor tOgO = gmem_thr_copy_O.partition_D(gO(_, _, 0));

  __syncthreads();

  // NOTE: Copy to gmem

  // TODO: Review the purpose of these two copies
  Tensor tOrO = make_tensor<Element>(shape(tOgO));
  cute::copy(gmem_tiled_copy_O, tOsO, tOrO);

  flash::copy(gmem_tiled_copy_O, tOrO, tOgO);

  // NOTE: Write back lse
  Tensor gLSE = local_tile(LSE, make_tile(Int<kBlockM>{}), make_coord(m_block));
  Tensor caccO = make_identity_tensor(Shape<Int<kBlockM>, Int<kHeadDim>>{});    // (BLK_M,BLK_K) -> (blk_m,blk_k)
  Tensor taccOcO = thr_mma.partition_C(caccO);                           // (MMA,MMA_M,MMA_K)
  static_assert(decltype(size<0>(taccOcO))::value == 4);
  // Convert to ((2, 2), MMA_M, MMA_K) then take only the row indices.
  // TODO: Review this shape
  Tensor taccOcO_row = logical_divide(taccOcO, Shape<_2>{})(make_coord(0, _), _, 0);
  CUTE_STATIC_ASSERT_V(size(lse) == size(taccOcO_row));                     // MMA_M
  // TODO: Understand the logic here
  if (get<1>(taccOcO_row(0)) == 0) {
      #pragma unroll
      for (int mi = 0; mi < size(lse); ++mi) {
          const int row = get<0>(taccOcO_row(mi));
          gLSE(row) = lse(mi);
      }
  }
}

template<typename Kernel_traits, bool Is_causal>
void run_flash_fwd(Flash_fwd_params &params, hipStream_t stream) {
  // TODO: check if works: default stream = 0
  using Element = typename Kernel_traits::Element;
  using SmemLayoutQ = typename Kernel_traits::SmemLayoutQ;
  using SmemLayoutK = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutV = typename Kernel_traits::SmemLayoutKV;

  const int num_m_block =
      (params.q_seqlen + Kernel_traits::kBlockM - 1) / Kernel_traits::kBlockM;

  dim3 grid(num_m_block, params.bs * params.head, 1);
  dim3 block(Kernel_traits::kNThreads);

  int smem_size = int(sizeof(SharedStorage<Element, SmemLayoutQ, SmemLayoutK, SmemLayoutV>));

  auto kernel = &flash_attention_v2_cutlass_kernel<Kernel_traits, Is_causal, Flash_fwd_params>;
  if (smem_size >= 48 * 1024) {
      CUDA_ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
  }

  // TODO: stream
  kernel<<<grid, block, smem_size>>>(params);
}

template<typename T, int Headdim>
void run_flash_fwd_(Flash_fwd_params &params, hipStream_t stream);

template<typename T, int Headdim>
void run_flash_fwd_(Flash_fwd_params &params, hipStream_t stream) {
    BOOL_SWITCH(params.is_causal, Is_causal, [&] {
        // run_flash_fwd<Flash_fwd_kernel_traits<Headdim, /*kBlockM_=*/128, /*kBlockN_=*/128, /*kNWarps_=*/4, T>, Is_causal>(params, stream);

        // TODO: Combination of kBlockM and kBlockN
        run_flash_fwd<Flash_fwd_kernel_traits<Headdim, /*kBlockM_=*/64, /*kBlockN_=*/64, /*kNWarps_=*/4, T>, Is_causal>(params, stream);
    });
}

// Entry point of Flash Attention
void run_flash_attn_cutlass(Flash_fwd_params &params, hipStream_t stream) {
    // FP16_SWITCH yields elem_type namespace
    FP16_SWITCH(!params.is_bf16, [&] {
        // FWD_HEADDIM_SWITCH yields kHeadDim constexpr
        FWD_HEADDIM_SWITCH(params.dim, [&] {
            run_flash_fwd_<elem_type, kHeadDim>(params, stream);
        });
    });
}

std::vector<torch::Tensor> flash_attention_v2_cutlass(torch::Tensor q, torch::Tensor k,
                                      torch::Tensor v, bool is_causal = false, float softmax_scale=1) {

  CHECK_INPUT(q);
  CHECK_INPUT(k);
  CHECK_INPUT(v);

  // Batch size
  int bs = q.size(0);
  // Number of heads
  int head = q.size(1);
  // Sequence length
  int seqlen = q.size(2);
  // Dimension
  int dim = q.size(3);
  auto out = torch::empty_like(q);

  auto opts = q.options();
  auto softmax_lse = torch::empty({bs, head, seqlen}, opts.dtype(torch::kFloat32));

  Flash_fwd_params params;
  set_params_fprop(params, q, k, v, out,
      softmax_lse.data_ptr(), softmax_scale, is_causal);

  run_flash_attn_cutlass(params, 0);

  // Wait until kernel finishes.
  hipDeviceSynchronize();
  CUDA_ERROR_CHECK(hipGetLastError());

  return {out, softmax_lse};
}


template <typename Kernel_traits, bool Is_causal=false, typename Params, int maskM=64, int maskN=64>
__global__ void flash_attention_block_v2_cutlass_kernel(const Params params) {

  using namespace cute;

  // M block index
  const int m_block = blockIdx.x;

  // Batch size * head
  const int base_id = blockIdx.y;
  // The thread index.
  const int tidx = threadIdx.x;

  // TODO: Pass in generic type
  // NOTE: Small trick
  using Element = typename Kernel_traits::Element;
  using ElementAccum = typename Kernel_traits::ElementAccum;
  // using TiledMMA = typename Kernel_traits::MMA;
  using TiledMMA = typename Kernel_traits::TiledMma;
  using index_t = typename Kernel_traits::index_t;
  using SmemLayoutQ = typename Kernel_traits::SmemLayoutQ;
  using SmemLayoutK = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutV = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutVt = typename Kernel_traits::SmemLayoutVtransposed;
  using SmemLayoutVtNoSwizzle = typename Kernel_traits::SmemLayoutVtransposedNoSwizzle;

  constexpr int kNWarps = Kernel_traits::kNWarps;
  constexpr int kBlockM = Kernel_traits::kBlockM;
  constexpr int kBlockN = Kernel_traits::kBlockN;
  constexpr int kHeadDim = Kernel_traits::kHeadDim;

  // Shared memory.
  extern __shared__ char smem_[];
  using SharedStorage = SharedStorage<Element, SmemLayoutQ, SmemLayoutK, SmemLayoutV>;
  SharedStorage &shared_storage = *reinterpret_cast<SharedStorage *>(smem_);

  const int bs_head_offset = base_id * params.head_stride;
  // TODO: Add assert
  const int row_factor = maskM / kBlockM;
  const int col_factor = maskN / kBlockN;
  const int m_mask = m_block / row_factor;
  const int num_n_mask = cute::ceil_div(params.k_seqlen, maskN);
  const int num_n_block = cute::ceil_div(params.k_seqlen, kBlockN);

  // Add mask start pointer
   // int *blockmask_ptr = params.block_mask_ptr + (batch_idx * params.num_blocksparse_heads + mask_type - 1) * int(params.seqlen_q_rounded / m_block_dim) * int(params.seqlen_k_rounded / n_block_dim) + int(loop_step_idx / row_factor) * int(params.seqlen_k_rounded / n_block_dim);
  int *mask_ptr = params.block_mask_ptr + base_id * params.mask_head_stride + m_mask * num_n_mask;
  int mask_id = 0, nbi = mask_ptr[0] * col_factor;
//   printf("----------------------%d %d %d %d\n", m_block, num_n_block, nbi, mask_ptr[num_n_block - 1]);
//   for (int i = 0; i <  num_n_block; i++) {
//     printf("%d   ", mask_ptr[i]);
//   }
//   printf("\n");

  // Empty line
  if (nbi < 0) {
    return;
  }

  // TODO: Base offset for MHA
  // NOTE: Convert C pointer to Tensor for convenience
  Tensor Q = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.q_ptr) + bs_head_offset),
      make_shape(params.q_seqlen, Int<kHeadDim>{}),
      make_stride(Int<kHeadDim>{}, Int<1>{}));
  Tensor K = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.k_ptr) + bs_head_offset),
      make_shape(params.k_seqlen, Int<kHeadDim>{}),
      make_stride(Int<kHeadDim>{}, Int<1>{}));
  Tensor V = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.v_ptr) + bs_head_offset),
      make_shape(params.k_seqlen, Int<kHeadDim>{}),
      make_stride(Int<kHeadDim>{}, Int<1>{}));
  Tensor O = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.out_ptr) + bs_head_offset),
      make_shape(params.q_seqlen, Int<kHeadDim>{}),
      make_stride(Int<kHeadDim>{}, Int<1>{}));
  // TODO:
  Tensor LSE = make_tensor(
      make_gmem_ptr(reinterpret_cast<ElementAccum *>(params.softmax_lse_ptr) + base_id * params.q_seqlen),
      // Shape<Int<kBlockM>, Stride<_1>{}>{}, 
      make_shape(params.q_seqlen),
      make_stride(Int<1>{}));

  // Load Q, K, V blocks
  // (kBlockM, kHeadDim, num_tile_n)
  Tensor gQ = local_tile(Q, make_tile(Int<kBlockM>{}, Int<kHeadDim>{}), make_coord(m_block, _));

  // (kBlockN, kHeadDim, num_tile_n)
  // NOTE: Loading pipeline, initial loading of K and V
  Tensor gK = local_tile(K, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(nbi, _));
  Tensor gV = local_tile(V, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(nbi, _));

  // Get MMA abstraction
  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(tidx);

  // Construct SMEM tensors.
  Tensor sQ = make_tensor(make_smem_ptr(shared_storage.smem_q.data()), SmemLayoutQ{});
  Tensor sK = make_tensor(make_smem_ptr(shared_storage.smem_k.data()), SmemLayoutK{});
  Tensor sV = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutV{});

  // Tensor for V Transpose; used in GEMM-II.
  Tensor sVt = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutVt{});
  Tensor sVtNoSwizzle = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutVtNoSwizzle{});

  // NOTE: Copy abstraction
  // NOTE: QKV gmem -> smem copy abstraction
  typename Kernel_traits::GmemTiledCopyQKV gmem_tiled_copy_QKV;
  auto gmem_thr_copy_QKV = gmem_tiled_copy_QKV.get_thread_slice(tidx);

  // NOTE: Define gmem -> smem copy src, dst
  Tensor tQgQ = gmem_thr_copy_QKV.partition_S(gQ(_, _, 0));
  Tensor tQsQ = gmem_thr_copy_QKV.partition_D(sQ);
  Tensor tKgK = gmem_thr_copy_QKV.partition_S(gK(_, _, 0));
  Tensor tKsK = gmem_thr_copy_QKV.partition_D(sK);
  Tensor tVgV = gmem_thr_copy_QKV.partition_S(gV(_, _, 0));
  Tensor tVsV = gmem_thr_copy_QKV.partition_D(sV);

  // NOTE: Define smem -> reg copy dst
  // partition_fragment is similar to partition, but returns a register representation
  Tensor tSrQ  = thr_mma.partition_fragment_A(sQ);                           // (MMA,MMA_M,MMA_K)
  Tensor tSrK  = thr_mma.partition_fragment_B(sK);                           // (MMA,MMA_N,MMA_K)
  Tensor tOrVt  = thr_mma.partition_fragment_B(sVtNoSwizzle);                // (MMA, MMA_K,MMA_N)

  //
  // Copy Atom retiling
  //

  // TODO: Understand atom retiling here

  // NOTE: Prepare the copy objects to copy Q, K, V to smem
  auto smem_tiled_copy_Q = make_tiled_copy_A(typename Kernel_traits::SmemCopyAtom{}, tiled_mma);
  auto smem_thr_copy_Q = smem_tiled_copy_Q.get_thread_slice(tidx);
  Tensor tSsQ = smem_thr_copy_Q.partition_S(sQ);

  auto smem_tiled_copy_K = make_tiled_copy_B(typename Kernel_traits::SmemCopyAtom{}, tiled_mma);
  auto smem_thr_copy_K = smem_tiled_copy_K.get_thread_slice(tidx);
  Tensor tSsK = smem_thr_copy_K.partition_S(sK);

  // TODO: Transpose during copy
  // NOTE: smem->reg copy of Vt
  auto smem_tiled_copy_V = make_tiled_copy_B(typename Kernel_traits::SmemCopyAtomTransposed{}, tiled_mma);
  auto smem_thr_copy_V = smem_tiled_copy_V.get_thread_slice(tidx);
  Tensor tOsVt = smem_thr_copy_V.partition_S(sVt);

  // Pipeline load initial Q, K
  // Load Q to smem
  flash::copy(gmem_tiled_copy_QKV, tQgQ, tQsQ);
  // Load K to smem
  flash::copy(gmem_tiled_copy_QKV, tKgK, tKsK);
  // Start asynchronous copy
  cute::cp_async_fence();

  Tensor rAccOut = partition_fragment_C(tiled_mma, Shape<Int<kBlockM>, Int<kHeadDim>>{});

  // step1: slice-k compute QK block
  // Q[BLOCK_M, BLOCK_N] @ K[BLOCK_M, BLOCK_N].T = O[BLOCK_M, BLOCK_M]
  //
  // step2:
  // advance K, V

  // NOTE: Number of K, V blocks: processing range
  const int n_block_min = 0;
  // NOTE: 1. mask between N BLOCKs if in causal mode
  int seqlen_start = m_block * kBlockM;
  int seqlen_end = (m_block + 1) * kBlockM;
  int n_block_max = Is_causal ? cute::ceil_div(seqlen_end, kBlockN) : cute::ceil_div(params.k_seqlen, kBlockN);

  // NOTE: Maximum values to record
  Tensor scores_max = make_tensor<ElementAccum>(Shape<Int<2 * size<1>(rAccOut)>>{});
  // NOTE: Denominator values to record
  Tensor scores_sum = make_fragment_like(scores_max);

  clear(rAccOut);
  
  while (nbi >= 0) {
    auto rAccScore = partition_fragment_C(tiled_mma, make_shape(Int<kBlockM>{}, Int<kBlockN>{}));

    clear(rAccScore);

    // Wait for the gmem -> smem copy of Q and K to complete, meaning Q and K are ready
    // wait<0> indicates waiting for 0 unfinished tasks

    flash::cp_async_wait<0>();
    __syncthreads();

    // Asynchronous loading of V during GEMM
    gV = local_tile(V, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(nbi, _));
    tVgV = gmem_thr_copy_QKV.partition_S(gV(_, _, 0));
    // Asynchronously load V into smem
    flash::copy(gmem_tiled_copy_QKV, tVgV, tVsV);
    // Initiate asynchronous copy
    cute::cp_async_fence();

    // O = Q@K.T
    // NOTE: Load data from smem to registers and perform gemm, **retile during loading**
    flash::gemm_smem(rAccScore, tSrQ, tSrK, tSsQ, tSsK, tiled_mma, smem_tiled_copy_Q, smem_tiled_copy_K,
        smem_thr_copy_Q, smem_thr_copy_K
    );

    Tensor scores = make_tensor(rAccScore.data(), flash::convert_layout_acc_rowcol(rAccScore.layout()));

    // NOTE: 2. Mask within N BLOCKs
    if (Is_causal == true && nbi * kBlockN >= seqlen_start) {
      flash::mask_within_nblock<kBlockM, kBlockN, kNWarps>(scores, m_block, nbi);
    }

    // NOTE: Wait for V loading to complete, prepare initial state for the next K load
    flash::cp_async_wait<0>();
    __syncthreads();

    // Advance K
    mask_id++;
    if (mask_id == num_n_block) {
      nbi = -1;
    }
    else if(mask_id % col_factor == 0) {
      // Load next mask
      nbi = mask_ptr[mask_id / col_factor] * col_factor;
    } else {
      nbi++;
    }

    if (nbi >= 0) {
      gK = local_tile(K, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(nbi, _));
      tKgK = gmem_thr_copy_QKV.partition_S(gK(_, _, 0));
      flash::copy(gmem_tiled_copy_QKV, tKgK, tKsK);
      cute::cp_async_fence();
    }

    // Compute softmax
    // NOTE: rAccOut records all the numerators after softmax
    mask_id == 0 ? flash::softmax_rescale_o</*Is_first=*/true>(scores, scores_max, scores_sum, rAccOut, params.softmax_scale) :
      flash::softmax_rescale_o</*Is_first=*/false>(scores, scores_max, scores_sum, rAccOut, params.softmax_scale);

    // Actual QK @ V execution
    // (score AKA rAccScore): QK[M, N] @ V[N, dim]
    // NOTE: DABC: F32F16F16F32, convert D type(F32) to A type(F16)
    // TODO: convert_type is currently hardcoded
    Tensor rP = flash::convert_type_f32_to_f16(rAccScore);
    // NOTE: Convert from layout C to layout A
    Tensor tOrP = make_tensor(rP.data(), flash::convert_layout_rowcol_Aregs<TiledMMA>(scores.layout()));

    flash::gemm_A_in_regs(rAccOut, tOrP, tOrVt, tOsVt, tiled_mma, smem_tiled_copy_V, smem_thr_copy_V);
  }

  // Epilogue

  // NOTE: Finally divide by the denominator
  // Reshape acc_o from (MMA=4, MMA_M, MMA_K) to (nrow=(2, MMA_M), ncol=(2, MMA_K))
  // AKA reshape to (nrow, ncol) but with specific MMA layout
  Tensor acc_o_rowcol = make_tensor(rAccOut.data(), flash::convert_layout_acc_rowcol(rAccOut.layout()));
  // NOTE: Save lse for backward
  Tensor lse = make_fragment_like(scores_sum);
  // for row
  #pragma unroll
  for (int mi = 0; mi < size<0>(acc_o_rowcol); ++mi) {
    float sum = scores_sum(mi);
    float inv_sum = (sum == 0.f || sum != sum) ? 1.f : 1.f / sum;
    // compute lse
    // NOTE: here we use max * scale 
    lse(mi) = (sum == 0.f || sum != sum) ? INFINITY : scores_max(mi) * params.softmax_scale + __logf(sum);
    float scale = inv_sum;
    // for col
    #pragma unroll
    for (int ni = 0; ni < size<1>(acc_o_rowcol); ++ni) {
      acc_o_rowcol(mi, ni) *= scale;
    }
  }

  // Convert acc_o from fp32 to fp16/bf16
  Tensor rO = flash::convert_type_f32_to_f16(rAccOut);
  // Reuse sQ's smem for sO copy out
  Tensor sO = make_tensor(sQ.data(), typename Kernel_traits::SmemLayoutO{});    // (SMEM_M,SMEM_N)

  // Partition sO to match the accumulator partitioning
  // TODO: review
  auto smem_tiled_copy_O = make_tiled_copy_C(typename Kernel_traits::SmemCopyAtomO{}, tiled_mma);
  auto smem_thr_copy_O = smem_tiled_copy_O.get_thread_slice(tidx);
  Tensor taccOrO = smem_thr_copy_O.retile_S(rO);        // ((Atom,AtomNum), MMA_M, MMA_N)
  Tensor taccOsO = smem_thr_copy_O.partition_D(sO);     // ((Atom,AtomNum),PIPE_M,PIPE_N)

  // NOTE: Copy to smem first
  cute::copy(smem_tiled_copy_O, taccOrO, taccOsO);

  Tensor gO = local_tile(O, make_tile(Int<kBlockM>{}, Int<kHeadDim>{}), make_coord(m_block, _));

  // Create a copy from smem to gmem
  typename Kernel_traits::GmemTiledCopyO gmem_tiled_copy_O;
  auto gmem_thr_copy_O = gmem_tiled_copy_O.get_thread_slice(tidx);
  Tensor tOsO = gmem_thr_copy_O.partition_S(sO);        // ((Atom,AtomNum),ATOM_M,ATOM_N)
  Tensor tOgO = gmem_thr_copy_O.partition_D(gO(_, _, 0));

  __syncthreads();

  // NOTE:: Copy again to gmem

  // TODO: review, what is the purpose of these two copy operations?
  Tensor tOrO = make_tensor<Element>(shape(tOgO));
  cute::copy(gmem_tiled_copy_O, tOsO, tOrO);

  flash::copy(gmem_tiled_copy_O, tOrO, tOgO);

  // NOTE: Write back lse
  Tensor gLSE = local_tile(LSE, make_tile(Int<kBlockM>{}), make_coord(m_block));
  Tensor caccO = make_identity_tensor(Shape<Int<kBlockM>, Int<kHeadDim>>{});    // (BLK_M,BLK_K) -> (blk_m,blk_k)
  Tensor taccOcO = thr_mma.partition_C(caccO);                           // (MMA,MMA_M,MMA_K)
  static_assert(decltype(size<0>(taccOcO))::value == 4);
  // Convert to ((2, 2), MMA_M, MMA_K) then take only the row indices.
  // TODO: review this shape
  Tensor taccOcO_row = logical_divide(taccOcO, Shape<_2>{})(make_coord(0, _), _, 0);
  CUTE_STATIC_ASSERT_V(size(lse) == size(taccOcO_row));                     // MMA_M
  // TODO: Clarify the logic here
  if (get<1>(taccOcO_row(0)) == 0) {
      #pragma unroll
      for (int mi = 0; mi < size(lse); ++mi) {
          const int row = get<0>(taccOcO_row(mi));
          // if (row < binfo.actual_seqlen_q - m_block * kBlockM) { gLSE(row) = lse(mi); }
          gLSE(row) = lse(mi);
      }
  }
}

template<typename Kernel_traits, bool Is_causal>
void run_flash_block_fwd(Block_flash_fwd_params &params, hipStream_t stream) {
  // TODO: check if works: default stream = 0
  using Element = typename Kernel_traits::Element;
  using SmemLayoutQ = typename Kernel_traits::SmemLayoutQ;
  using SmemLayoutK = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutV = typename Kernel_traits::SmemLayoutKV;

  const int num_m_block =
      (params.q_seqlen + Kernel_traits::kBlockM - 1) / Kernel_traits::kBlockM;

  dim3 grid(num_m_block, params.bs * params.head, 1);
  dim3 block(Kernel_traits::kNThreads);

  int smem_size = int(sizeof(SharedStorage<Element, SmemLayoutQ, SmemLayoutK, SmemLayoutV>));

  auto kernel = &flash_attention_block_v2_cutlass_kernel<Kernel_traits, Is_causal, Block_flash_fwd_params, 64, 64>;
  // NOTE: When smem is too large, need to set this
  if (smem_size >= 48 * 1024) {
      CUDA_ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
  }

  // TODO: stream
  kernel<<<grid, block, smem_size>>>(params);
}

template<typename T, int Headdim>
void run_flash_block_fwd_(Block_flash_fwd_params &params, hipStream_t stream);

// TODO: Write specialization for each case, currently using general template
// For example, run_flash_fwd_hdim32 for specialization with hdim=32
// This allows adjustment of kBlockN and kBlockM combinations for better compilation speed
template<typename T, int Headdim>
void run_flash_block_fwd_(Block_flash_fwd_params &params, hipStream_t stream) {
    BOOL_SWITCH(params.is_causal, Is_causal, [&] {
        // run_flash_fwd<Flash_fwd_kernel_traits<Headdim, /*kBlockM_=*/128, /*kBlockN_=*/128, /*kNWarps_=*/4, T>, Is_causal>(params, stream);

        // TODO: Adjust kBlockM, kBlockN combinations
        run_flash_block_fwd<Flash_fwd_kernel_traits<Headdim, /*kBlockM_=*/64, /*kBlockN_=*/64, /*kNWarps_=*/4, T>, Is_causal>(params, stream);
    });
}

// Entry point for flash attention
void run_flash_attn_block_cutlass(Block_flash_fwd_params &params, hipStream_t stream) {
    // FP16_SWITCH yield elem_type namespace
    FP16_SWITCH(!params.is_bf16, [&] {
        // FWD_HEADDIM_SWITCH yield kHeadDim constexpr
        FWD_HEADDIM_SWITCH(params.dim, [&] {
            run_flash_block_fwd_<elem_type, kHeadDim>(params, stream);
        });
    });
}


std::vector<torch::Tensor> flash_attention_block_v2_cutlass(torch::Tensor q, torch::Tensor k,
                                      torch::Tensor v, torch::Tensor row_mask, bool is_causal = false, float softmax_scale=1) {

  CHECK_INPUT(q);
  CHECK_INPUT(k);
  CHECK_INPUT(v);

  // Batch size
  int bs = q.size(0);
  // Number of heads
  int head = q.size(1);
  // Sequence length
  int seqlen = q.size(2);
  // Dimension
  int dim = q.size(3);
  auto out = torch::empty_like(q);

  auto opts = q.options();
  auto softmax_lse = torch::empty({bs, head, seqlen}, opts.dtype(torch::kFloat32));

  Block_flash_fwd_params params;
  set_params_fprop(params, q, k, v, out,
      softmax_lse.data_ptr(), softmax_scale, is_causal);

  // TODO: get ptr
  params.block_mask_ptr = reinterpret_cast<int*>(row_mask.data_ptr());
  params.mask_head_stride = row_mask.stride(1);

  run_flash_attn_block_cutlass(params, 0);

  // Wait until kernel finishes
  hipDeviceSynchronize();
  CUDA_ERROR_CHECK(hipGetLastError());

  return {out, softmax_lse};
}